#include "hip/hip_runtime.h"
#include "nsc_solver.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>

// CUDA kernel for initializing phase field
__global__ void initializeFieldsKernel(double* phi, double* u, double* v, int gridSize, double bubbleRadius) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < gridSize && j < gridSize) {
        double x = i - gridSize / 2;
        double y = j - gridSize / 2;
        phi[i * gridSize + j] = (x * x + y * y <= bubbleRadius * bubbleRadius) ? 1.0 : -1.0;
        u[i * gridSize + j] = 0.0;
        v[i * gridSize + j] = 0.0;
    }
}

// Kernel for updating the chemical potential
__global__ void updateChemicalPotentialKernel(double* phi, double* mu, SimulationParameters params, int gridSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < gridSize - 1 && j > 0 && j < gridSize - 1) {
        double laplacian = phi[(i + 1) * gridSize + j] + phi[(i - 1) * gridSize + j] +
                           phi[i * gridSize + (j + 1)] + phi[i * gridSize + (j - 1)] -
                           4.0 * phi[i * gridSize + j];
        mu[i * gridSize + j] = -phi[i * gridSize + j] + phi[i * gridSize + j] * phi[i * gridSize + j] * phi[i * gridSize + j] - 
                               params.epsilon * params.epsilon * laplacian;
    }
}

// Kernel for updating phase field
__global__ void updatePhaseFieldKernel(double* phi, double* mu, SimulationParameters params, int gridSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < gridSize - 1 && j > 0 && j < gridSize - 1) {
        double muLaplacian = mu[(i + 1) * gridSize + j] + mu[(i - 1) * gridSize + j] +
                             mu[i * gridSize + (j + 1)] + mu[i * gridSize + (j - 1)] -
                             4.0 * mu[i * gridSize + j];
        phi[i * gridSize + j] += params.dt * params.mobility * muLaplacian;
    }
}

// Kernel for updating velocity field (dummy implementation for now)
__global__ void updateVelocityFieldKernel(double* u, double* v, double* phi, SimulationParameters params, int gridSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < gridSize && j < gridSize) {
        u[i * gridSize + j] = 0.0;
        v[i * gridSize + j] = 0.0;
    }
}

// Function to allocate and launch CUDA kernels
void initializeFieldsCUDA(double* d_phi, double* d_u, double* d_v, int gridSize, double bubbleRadius) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((gridSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (gridSize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeFieldsKernel<<<numBlocks, threadsPerBlock>>>(d_phi, d_u, d_v, gridSize, bubbleRadius);
    hipDeviceSynchronize();
}

void updateChemicalPotentialCUDA(double* d_phi, double* d_mu, SimulationParameters params, int gridSize) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((gridSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (gridSize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    updateChemicalPotentialKernel<<<numBlocks, threadsPerBlock>>>(d_phi, d_mu, params, gridSize);
    hipDeviceSynchronize();
}

void updatePhaseFieldCUDA(double* d_phi, double* d_mu, SimulationParameters params, int gridSize) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((gridSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (gridSize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    updatePhaseFieldKernel<<<numBlocks, threadsPerBlock>>>(d_phi, d_mu, params, gridSize);
    hipDeviceSynchronize();
}

void updateVelocityFieldCUDA(double* d_u, double* d_v, double* d_phi, SimulationParameters params, int gridSize) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((gridSize + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (gridSize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    updateVelocityFieldKernel<<<numBlocks, threadsPerBlock>>>(d_u, d_v, d_phi, params, gridSize);
    hipDeviceSynchronize();
}
void saveResultsFromGPU(const double* d_phi, int gridSize, int step, const std::string& filename) {
    std::vector<double> h_phi(gridSize * gridSize);
    hipMemcpy(h_phi.data(), d_phi, gridSize * gridSize * sizeof(double), hipMemcpyDeviceToHost);

    std::ofstream outFile(filename + "_step_" + std::to_string(step) + ".csv");
    if (!outFile) {
        std::cerr << "Error: Unable to open file " << filename << " for writing!" << std::endl;
        return;
    }

    for (int i = 0; i < gridSize; ++i) {
        for (int j = 0; j < gridSize; ++j) {
            outFile << h_phi[i * gridSize + j];
            if (j < gridSize - 1) outFile << ",";
        }
        outFile << "\n";
    }
    outFile.close();
}

